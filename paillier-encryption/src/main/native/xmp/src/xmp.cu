#include "hip/hip_runtime.h"
/***
Copyright (c) 2015, NVIDIA CORPORATION.  All rights reserved.

Permission is hereby granted, free of charge, to any person obtaining a 
copy of this software and associated documentation files (the "Software"), 
to deal in the Software without restriction, including without limitation 
the rights to use, copy, modify, merge, publish, distribute, sublicense, 
and/or sell copies of the Software, and to permit persons to whom the 
Software is furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in 
all copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR 
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, 
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE 
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER 
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING 
FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS 
IN THE SOFTWARE.
***/
#include "xmp_internal.h"
#include <thrust/scan.h>
#include <thrust/unique.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h> 

void *xmpDeviceMalloc(size_t bytes) {
  void* retval;
  if(hipSuccess!=hipMalloc(&retval,bytes))
    return 0;
  return retval;
}
void xmpDeviceFree(void *ptr) {
  hipFree(ptr);
}

//create xmp_handle
xmpError_t XMPAPI xmpHandleCreate(xmpHandle_t *handle) {
  return xmpHandleCreateWithMemoryFunctions(handle,malloc,free,xmpDeviceMalloc,xmpDeviceFree);
}

xmpError_t XMPAPI xmpHandleCreateWithMemoryFunctions(xmpHandle_t *handle,xmpAllocFunc ha, xmpFreeFunc hf, xmpAllocFunc da, xmpFreeFunc df) {
  XMP_CHECK_NE(handle,NULL);
  
  if(ha==NULL) ha=malloc;
  if(hf==NULL) hf=free;
  if(da==NULL) da=xmpDeviceMalloc;
  if(df==NULL) df=xmpDeviceFree;

  *handle=(_xmpHandle_t*)ha(sizeof(_xmpHandle_t));

  if(*handle==0)
    return xmpErrorInvalidMalloc;

  (*handle)->stream=0;
  (*handle)->scratch=NULL;
  (*handle)->scratchSize=0;
  (*handle)->tmpOutSize=0;
  (*handle)->tmpOut=NULL;
  (*handle)->ha=ha;
  (*handle)->hf=hf;
  (*handle)->da=da;
  (*handle)->df=df;

  (*handle)->policy=const_cast<xmpExecutionPolicy_t>(&xmpDefaultExecutionPolicy);

  if(hipSuccess!=hipGetDevice(&((*handle)->device)))
    return xmpErrorInvalidDevice;

  //verify device properties
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop,(*handle)->device);

  if(prop.major<2)
    return xmpErrorInvalidDevice;

  (*handle)->memorySize=prop.totalGlobalMem;
  (*handle)->arch=prop.major*10+prop.minor;
  (*handle)->smCount=prop.multiProcessorCount;

  XMP_CHECK_CUDA();
  return xmpErrorSuccess;
}


//destroy xmp_handle
xmpError_t XMPAPI xmpHandleDestroy(xmpHandle_t handle) {
  XMP_SET_DEVICE(handle);

  handle->df(handle->scratch);
  handle->df(handle->tmpOut);

  //free handle
  handle->hf(handle); 
  
  if(hipSuccess!=hipPeekAtLastError())
    return xmpErrorCuda;

  XMP_CHECK_CUDA();
  return xmpErrorSuccess;
}

//increases scratch size if necessary
xmpError_t xmpSetNecessaryScratchSize(xmpHandle_t handle, size_t bytes) {
  size_t maxSize=0;
  
  XMP_SET_DEVICE(handle);

  if(handle->policy!=NULL)
    maxSize=handle->policy->scratch_size_limit;
  if(maxSize==0)
    maxSize=handle->memorySize/4*3;
  if(bytes>maxSize)
    return xmpErrorIncreaseScratchLimit;
    
  if(handle->scratchSize<bytes)  {
    if(handle->scratch!=0) 
      //free existing scratch
      handle->df(handle->scratch);
    //allocate scratch
    handle->scratch=handle->da(bytes);
    handle->scratchSize=bytes;
  }
  if(handle->scratch==0)
    return xmpErrorInvalidCudaMalloc;
  XMP_CHECK_CUDA();
  return xmpErrorSuccess;
}
//increases output size if necessary
xmpError_t xmpSetNecessaryOutSize(xmpHandle_t handle, size_t bytes) {
  size_t maxSize=0;
  
  XMP_SET_DEVICE(handle);

  if(handle->tmpOutSize<bytes)  {
    if(handle->tmpOut!=0) 
      //free existing scratch
      handle->df(handle->tmpOut);
    //allocate scratch
    handle->tmpOut=handle->da(bytes);
    handle->tmpOutSize=bytes;
  }
  if(handle->scratch==0)
    return xmpErrorInvalidCudaMalloc;
  XMP_CHECK_CUDA();
  return xmpErrorSuccess;
}

//get memory functions
xmpError_t XMPAPI xmpHandleGetMemoryFunctions(xmpHandle_t handle, xmpAllocFunc *ha, xmpFreeFunc *hf, xmpAllocFunc *da, xmpFreeFunc *df) {
  if(ha!=NULL) *ha=handle->ha;
  if(hf!=NULL) *hf=handle->hf;
  if(da!=NULL) *da=handle->da;
  if(hf!=NULL) *df=handle->df;

  XMP_CHECK_CUDA();
  return xmpErrorSuccess;
}

//set stream for CUDA operations
xmpError_t XMPAPI xmpHandleSetStream(xmpHandle_t handle, hipStream_t stream) {
  //TODO check that the stream and handle device match (not supported in CUDA yet)

  //TODO set dependcy between old stream and new stream
  handle->stream=stream;

  XMP_CHECK_CUDA();
  return xmpErrorSuccess;
}


//get the stream associated with the handle
xmpError_t XMPAPI xmpHandleGetStream(xmpHandle_t handle, hipStream_t *stream) {
  XMP_CHECK_NE(stream,NULL);

  *stream=handle->stream;

  XMP_CHECK_CUDA();
  return xmpErrorSuccess;
}

//get the device associated with the handle
xmpError_t XMPAPI xmpHandleGetDevice(xmpHandle_t handle, int32_t *device) {
  XMP_CHECK_NE(device,NULL);

  *device=handle->device;

  XMP_CHECK_CUDA();
  return xmpErrorSuccess;
}

xmpError_t XMPAPI xmpHandleSetExecutionPolicy(xmpHandle_t handle, xmpExecutionPolicy_t policy) {
  if(policy!=NULL)
    handle->policy=policy;
  else
    handle->policy=const_cast<xmpExecutionPolicy_t>(&xmpDefaultExecutionPolicy);

  return xmpErrorSuccess;
}

//creates an execution policy
xmpError_t XMPAPI xmpExecutionPolicyCreate(xmpHandle_t handle, xmpExecutionPolicy_t *policy) {
  XMP_CHECK_NE(policy,NULL);
  XMP_SET_DEVICE(handle);
  
  //allocate policy
  *policy=( _xmpExecutionPolicy_t*)handle->ha(sizeof(_xmpExecutionPolicy_t));
  if(*policy==0)
    return xmpErrorInvalidMalloc;
 
  (*policy)->scratch_size_limit=0;
  for(int i=0;i<XMP_EXECUTION_POLICY_MAX_INDICES_ARRAYS;i++)
    (*policy)->indices[i]=NULL;

  (*policy)->algorithm=xmpAlgorithmDefault;

  return xmpErrorSuccess;
}
//destroys an execution policy
xmpError_t XMPAPI xmpExecutionPolicyDestroy(xmpHandle_t handle, xmpExecutionPolicy_t policy) {
  XMP_SET_DEVICE(handle);

  //free indices
  for(int i=0;i<XMP_EXECUTION_POLICY_MAX_INDICES_ARRAYS;i++) {
    if(policy->indices[i]!=NULL) 
      handle->df(policy->indices[i]);
  }

  //free handle
  handle->hf(policy);

  XMP_CHECK_CUDA();
  return xmpErrorSuccess;
}

//set dynamic indices
xmpError_t XMPAPI xmpExecutionPolicySetIndicesAsync(xmpHandle_t handle, xmpExecutionPolicy_t policy, uint32_t which_integer, uint32_t *indices, uint32_t count) {
  XMP_SET_DEVICE(handle);
  
  
  if(which_integer>=XMP_EXECUTION_POLICY_MAX_INDICES_ARRAYS) {
    return xmpErrorInvalidParameter;
  }


  uint32_t **pindices=&policy->indices[which_integer];
  uint32_t *pcount=&policy->indices_count[which_integer];

  //if the parameter is null clear existing indices
  if(indices==NULL) {
    if(*pindices!=NULL) {
      handle->df(*pindices);
      *pindices=NULL;
    }
  } else {
    //if already allocated but a different size
    if(*pindices!=NULL && *pcount!=count) {
      //free old indices
      handle->df(*pindices);
      *pindices=NULL;
    }

    //if not allocated
    if(*pindices==NULL) {
      //allocate new indices
      *pindices=(uint32_t*)handle->da(sizeof(uint32_t)*count);
      *pcount=count;
      if(*pindices==NULL)  
        return xmpErrorInvalidCudaMalloc;
    }

    hipMemcpyAsync(*pindices,indices,sizeof(uint32_t)*count,hipMemcpyDefault,handle->stream);
  
    *pcount=count;
  }


  return xmpErrorSuccess;
}
  
xmpError_t XMPAPI xmpExecutionPolicySetIndices(xmpHandle_t handle, xmpExecutionPolicy_t policy, uint32_t which_integer, uint32_t *indices, uint32_t count) {
  xmpError_t error=xmpExecutionPolicySetIndicesAsync(handle,policy,which_integer,indices,count);
  hipStreamSynchronize(handle->stream);
  return error;
}

xmpError_t XMPAPI xmpExecutionPolicySetParameter(xmpHandle_t handle, xmpExecutionPolicy_t policy, xmpExecutionPolicyParam_t param, xmpExecutionPolicyValue_t val) {
  switch(param) {
    case xmpAlgorithm:
      policy->algorithm=val.algorithm;
      break;
    case xmpScratchSizeLimit:
      policy->scratch_size_limit=val.size;
      break;
    default:
      return xmpErrorInvalidParameter;
  }
  return xmpErrorSuccess;
}

xmpError_t XMPAPI xmpExecutionPolicyGetParameter(xmpHandle_t handle, xmpExecutionPolicy_t policy, xmpExecutionPolicyParam_t param, xmpExecutionPolicyValue_t &val) {
  switch(param) {
    case xmpAlgorithm:
      val.algorithm=policy->algorithm;
      break;
    case xmpScratchSizeLimit:
      val.size=policy->scratch_size_limit;
      break;
    default:
      return xmpErrorInvalidParameter;
  }
  return xmpErrorSuccess;
}

//allocate array of integers
xmpError_t XMPAPI xmpIntegersCreate(xmpHandle_t handle, xmpIntegers_t *x, uint32_t precision, uint32_t count) {
  XMP_CHECK_NE(x,NULL);
  XMP_SET_DEVICE(handle);
 
  if(count==0 || count>0xfffffffc)
    return xmpErrorInvalidCount;

  //allocate integer handle
  *x=(_xmpIntegers_t*)handle->ha(sizeof(_xmpIntegers_t));
  
  if(*x==0)
    return xmpErrorInvalidMalloc;

  uint32_t bits_per_limb=sizeof(xmpLimb_t)*8;
  
  if(precision%(sizeof(uint32_t)*8)!=0)
    return xmpErrorUnsupported;

  //precision=ROUND_UP(precision,bits_per_limb);
  uint32_t stride=ROUND_UP(count,128/sizeof(xmpLimb_t));  //round up to 128 byte boundaries
  uint32_t nlimbs=precision/bits_per_limb;
   
  (*x)->count=count;
  (*x)->precision=precision;
  (*x)->nlimbs=nlimbs;
  (*x)->device=handle->device;
  (*x)->format=xmpFormatNone;
  (*x)->stride=stride;

  //allocate array of integers on the device
  (*x)->climbs=(xmpLimb_t*)handle->da(sizeof(xmpLimb_t)*nlimbs*count);
  (*x)->slimbs=(xmpLimb_t*)handle->da(sizeof(xmpLimb_t)*nlimbs*stride);
  if((*x)->climbs==0 || (*x)->slimbs==0 )
    return xmpErrorInvalidCudaMalloc;

  XMP_CHECK_CUDA();
  return xmpErrorSuccess;
}

//free an array of integers
xmpError_t XMPAPI xmpIntegersDestroy(xmpHandle_t handle, xmpIntegers_t x) {
  XMP_SET_DEVICE(handle);

  //free array of integers using hipFree here
  handle->df(x->climbs);
  handle->df(x->slimbs);

  //free integer handle
  handle->hf(x);

  XMP_CHECK_CUDA();
  return xmpErrorSuccess;
}

//get the device associated with the handle
xmpError_t XMPAPI xmpIntegersGetPrecision(xmpHandle_t handle, xmpIntegers_t x, uint32_t *precision) {
  XMP_CHECK_NE(precision,NULL);

  *precision=x->precision;

  XMP_CHECK_CUDA();
  return xmpErrorSuccess;
}
//get the device associated with the handle
xmpError_t XMPAPI xmpIntegersGetCount(xmpHandle_t handle, xmpIntegers_t x, uint32_t *count) {
  XMP_CHECK_NE(count,NULL);

  *count=x->count;

  XMP_CHECK_CUDA();
  return xmpErrorSuccess;
}

int32_t query_endianess() {
  int32_t num=1;
  if(*(char *)&num == 1)
    return -1;
  else
    return 1;
}

__device__ inline uint8_t byte_swap(uint8_t w) { return w; }
__device__ inline uint16_t byte_swap(uint16_t w) { 
  asm("{"
      ".reg .b8 %wlo; \n"
      ".reg .b8 %whi; \n"
      "mov.b16 {%wlo,%whi}, %0; \n"
      "mov.b16 %0, {%whi,%wlo}; \n"
      "}" : "+h"(w) );
  return w; 
}
__device__ inline uint32_t byte_swap(uint32_t w) { 
  asm("{" 
    "prmt.b32    %0, %0, %0, 0x0123; \n"
      "}" : "+r"(w) );  
  return w; 
}
__device__ inline uint64_t byte_swap(uint64_t w) {
  asm("{"
    ".reg .u32 %alo; \n"
    ".reg .u32 %ahi; \n"
    ".reg .u32 %blo; \n"
    ".reg .u32 %bhi; \n"
    "mov.b64     {%alo,%ahi}, %0;   \n"
    "prmt.b32    %bhi, %alo, %ahi, 0x0123; \n"
    "prmt.b32    %blo, %alo, %ahi, 0x4567; \n"
    "mov.b64     %0,   {%blo,%bhi}; \n"
  "}" : "+l"(w) );
  return w; 
}

__global__ void printWordsStrided_kernel(xmpLimb_t* data, int limbs, int stride, int count) {
  for(int i=0;i<count;i++) {
    printf("i=%d\n    ",i);
    for(int j=limbs-1;j>=0;j--) {
      printf("%08x",data[j*stride+i]);
    }
    printf("\n");
  }
}

void printWordsStrided(xmpLimb_t* data, int limbs, int stride, int count) {
  printWordsStrided_kernel<<<1,1>>>(data,limbs,stride,count);
  hipDeviceSynchronize();
}

__global__ void printWordsCompact_kernel(xmpLimb_t* data, int limbs, int count) {
  for(int i=0;i<count;i++) {
    printf("i=%d\n    ",i);
    for(int j=limbs-1;j>=0;j--) {
      printf("%08x",data[i*limbs+j]);
    }
    printf("\n");
  }
}

void printWordsCompact(xmpLimb_t* data, int limbs, int count) {
  printWordsCompact_kernel<<<1,1>>>(data,limbs,count);
  hipDeviceSynchronize();
}



//transforms an array of data.  Can reverse the order, endian, and zero out the top nails bits of each word.
template<class word_t>
__global__ void xmpTransform(word_t *output, word_t *input, uint32_t count, uint32_t words, int32_t order, int32_t endian, uint32_t nails, uint32_t *out_indices, uint32_t *in_indices, uint32_t out_indices_count, uint32_t in_indices_count) {
  for(uint32_t i=blockIdx.y*blockDim.y+threadIdx.y;i<count;i+=blockDim.y*gridDim.y) {
    for(uint32_t j=blockIdx.x*blockDim.x+threadIdx.x;j<words;j+=blockDim.x*gridDim.x) {
      
      uint32_t in_idx=i, out_idx=i;

      if(NULL!=out_indices) out_idx=out_indices[out_idx%out_indices_count];
      if(NULL!=in_indices) in_idx=in_indices[in_idx%in_indices_count];

      //Read in the order we want to store
      uint32_t offset= (order==xmpNativeOrder) ? j : words-j-1;
      uint32_t idx=in_idx*words + offset;

      //read word
      word_t w=input[idx];
      
      //byte swap if big endian
      if(endian!=xmpNativeEndian) w=byte_swap(w);

      //apply nails
      word_t mask=word_t(-1)>>nails;
      w&=mask;
      
      //write in least significant first ordering
      output[out_idx*words+j]=w;
    }
  }
}


xmpError_t inline xmpIntegersImportInternal(xmpHandle_t handle, xmpIntegers_t out, uint32_t words, int32_t order, size_t size, int32_t endian, int32_t nails, void* in, uint32_t count, bool async) {
  XMP_CHECK_NE(in,NULL);

  xmpExecutionPolicy_t policy=handle->policy;

  //verify handle device and out device match
  int32_t device=handle->device;
  if(out->device!=device)
    return xmpErrorInvalidDevice;

  XMP_SET_DEVICE(handle);

  if(endian==0) endian=query_endianess();

  if(size!=1 && size!=2 && size!=4 && size!=8)
    return xmpErrorInvalidParameter;

  if(count==0 || count>out->count || words==0 || (order!=1 && order!=-1) ||  (endian!=1 && endian !=-1))
    return xmpErrorInvalidParameter;

  if(words*size*8!=out->precision)
    return xmpErrorInvalidPrecision;

  size_t bytes=count*words*size;
  if(endian==xmpNativeEndian && order==xmpNativeOrder && nails==0 && policy->indices[0]==NULL) {
    //common case, count & precision match, little endian, nails=0, no dynamic indexing,  no temporary memory needed just copy in
    if(hipSuccess!=hipMemcpyAsync(out->climbs,in,bytes,hipMemcpyDefault,handle->stream))
      return xmpErrorCuda;
  } else {
    //check if we know where this pointer came from, if not assume host
    hipPointerAttribute_t attrib;
    hipError_t error=hipPointerGetAttributes(&attrib,in);
    if(error!=hipSuccess) {
      if(error==hipErrorInvalidValue) {
        hipGetLastError();  //reset to hipSuccess
        attrib.memoryType=hipMemoryTypeHost;
      } else {
        return xmpErrorCuda;
      }
    }
    
    void* src=in;
    if(attrib.memoryType==hipMemoryTypeHost) {

      xmpError_t e=xmpSetNecessaryScratchSize(handle,bytes);
      if(e!=xmpErrorSuccess) return e;
      
      src=handle->scratch;

      //copy down to temporary memory
      if(hipSuccess!=hipMemcpyAsync(src,in,bytes,hipMemcpyDefault,handle->stream))
        return xmpErrorCuda;
    }

    //x = words
    //y = count
    dim3 blocks,threads;
    threads.x=MIN(words,128);           //Use 1 thread per word (max 128)
    threads.y=DIV_ROUND_UP(128,threads.x);  //block size = ~128 threads
    blocks.x=DIV_ROUND_UP(words,threads.x);
    blocks.y=DIV_ROUND_UP(count,threads.y);
 
    //unpack from temporary memory
    switch(size) {
      case 1:
        xmpTransform<<<blocks,threads,0,handle->stream>>>((uint8_t*)out->climbs,(uint8_t*)src,count,words,order,endian,nails,NULL,policy->indices[0],NULL,policy->indices_count[0]);
        break;
      case 2:
        xmpTransform<<<blocks,threads,0,handle->stream>>>((uint16_t*)out->climbs,(uint16_t*)src,count,words,order,endian,nails,NULL,policy->indices[0],NULL,policy->indices_count[0]);
        break;
      case 4:
        xmpTransform<<<blocks,threads,0,handle->stream>>>((uint32_t*)out->climbs,(uint32_t*)src,count,words,order,endian,nails,NULL,policy->indices[0],NULL,policy->indices_count[0]);
        break;
      case 8:
        xmpTransform<<<blocks,threads,0,handle->stream>>>((uint64_t*)out->climbs,(uint64_t*)src,count,words,order,endian,nails,NULL,policy->indices[0],NULL,policy->indices_count[0]);
        break;
      default:
        return xmpErrorInvalidParameter;
    };
  }
  out->setFormat(xmpFormatCompact);
  if(!async) hipStreamSynchronize(handle->stream);

  XMP_CHECK_CUDA();
  return xmpErrorSuccess;
}

xmpError_t XMPAPI xmpIntegersImport(xmpHandle_t handle, xmpIntegers_t out, uint32_t words, int32_t order, size_t size, int32_t endian, uint32_t nails, void* in, uint32_t count) {
  return xmpIntegersImportInternal(handle,out,words,order,size,endian,nails,in,count,false);
}
xmpError_t XMPAPI xmpIntegersImportAsync(xmpHandle_t handle, xmpIntegers_t out, uint32_t words, int32_t order, size_t size, int32_t endian, uint32_t nails, void* in, uint32_t count) {
  return xmpIntegersImportInternal(handle,out,words,order,size,endian,nails,in,count,true);
}
//export count integers of size bytes from in into out
xmpError_t inline xmpIntegersExportInternal(xmpHandle_t handle, void* out, uint32_t *words, int32_t order, size_t size, int32_t endian, uint32_t nails, xmpIntegers_t in, uint32_t count, bool async) {
  XMP_CHECK_NE(out,NULL);

  xmpExecutionPolicy_t policy=handle->policy;
  
  //verify handle device and in device match
  int32_t device=handle->device;
  if(in->device!=device)
    return xmpErrorInvalidDevice;
  
  XMP_SET_DEVICE(handle);

  if(endian==0) endian=query_endianess();
  
  if(size!=1 && size!=2 && size!=4 && size!=8)
    return xmpErrorInvalidParameter;

  if(count==0 || count>in->count || (order!=1 && order!=-1) ||  (endian!=1 && endian !=-1))
    return xmpErrorInvalidParameter;
  
  if(xmpErrorSuccess!=in->requireFormat(handle,xmpFormatCompact))
    return xmpErrorInvalidFormat;

  uint32_t limbs=in->nlimbs;
  uint32_t w = limbs * (uint32_t)sizeof(xmpLimb_t) / size;
  size_t bytes=count*limbs*sizeof(xmpLimb_t);
  if(endian==xmpNativeEndian && order==xmpNativeOrder && nails==0  && policy->indices[0]==NULL) {
    //common case, naitve endian and order, , nails=0, no dynamic indexing, no temporary memory needed just copy in
    if(hipSuccess!=hipMemcpyAsync(out,in->climbs,bytes,hipMemcpyDefault,handle->stream))
      return xmpErrorCuda;
  } else {

    //check if we know where this pointer came from, if not assume host
    hipPointerAttribute_t attrib;
    hipError_t error=hipPointerGetAttributes(&attrib,out);
    if(error!=hipSuccess) {
      if(error==hipErrorInvalidValue) {
        hipGetLastError();  //reset to hipSuccess
        attrib.memoryType=hipMemoryTypeHost;
      } else {
        return xmpErrorCuda;
      }
    }

    void* dst=out;
    if(attrib.memoryType==hipMemoryTypeHost) {
      xmpError_t e=xmpSetNecessaryScratchSize(handle,bytes);
      if(e!=xmpErrorSuccess) return e;
      dst=handle->scratch;
    }

    //x = words
    //y = count
    dim3 blocks,threads;
    threads.x=MIN(w,128);           //Use 1 thread per word (max 128)
    threads.y=DIV_ROUND_UP(128,threads.x);  //block size = ~128 threads
    blocks.x=DIV_ROUND_UP(w,threads.x);
    blocks.y=DIV_ROUND_UP(count,threads.y);

    //pack to temporary memory
    switch(size) {
      case 1:
        xmpTransform<<<blocks,threads,0,handle->stream>>>((uint8_t*)dst,(uint8_t*)in->climbs,count,w,order,endian,nails,policy->indices[0],NULL,policy->indices_count[0],NULL);
        break;
      case 2:
        xmpTransform<<<blocks,threads,0,handle->stream>>>((uint16_t*)dst,(uint16_t*)in->climbs,count,w,order,endian,nails,policy->indices[0],NULL,policy->indices_count[0],NULL);
        break;
      case 4:
        xmpTransform<<<blocks,threads,0,handle->stream>>>((uint32_t*)dst,(uint32_t*)in->climbs,count,w,order,endian,nails,policy->indices[0],NULL,policy->indices_count[0],NULL);
        break;
      case 8:
        xmpTransform<<<blocks,threads,0,handle->stream>>>((uint64_t*)dst,(uint64_t*)in->climbs,count,w,order,endian,nails,policy->indices[0],NULL,policy->indices_count[0],NULL);
        break;
      default:
        return xmpErrorInvalidParameter;
    };
   
    if(attrib.memoryType==hipMemoryTypeHost) {
      //copy up from temporary memory
      if(hipSuccess!=hipMemcpyAsync(out,dst,bytes,hipMemcpyDefault,handle->stream))
        return xmpErrorCuda;
    }
  }

  if(words!=0) *words=w;
  if(!async) hipStreamSynchronize(handle->stream);
  XMP_CHECK_CUDA();
  return xmpErrorSuccess;
}

xmpError_t XMPAPI xmpIntegersExport(xmpHandle_t handle, void* out, uint32_t *words, int32_t order, size_t size, int32_t endian, uint32_t nails, xmpIntegers_t in, uint32_t count) {
  return xmpIntegersExportInternal(handle,out,words,order,size,endian,nails,in,count,false);
}

xmpError_t XMPAPI xmpIntegersExportAsync(xmpHandle_t handle, void* out, uint32_t *words, int32_t order, size_t size, int32_t endian, uint32_t nails, xmpIntegers_t in, uint32_t count) {
  return xmpIntegersExportInternal(handle,out,words,order,size,endian,nails,in,count,true);
}


struct ReorderParams {
  uint32_t *out_limbs, *in_limbs;             //limb data
  uint32_t *out_indices, *in_indices;             //index arrays
  uint32_t out_indices_count, in_indices_count;   //index array count length
  uint32_t in_count;                          //actual length of the integer array
  uint32_t out_words, in_words;               //precision 
  uint32_t count;                             //number of elements to copy
};
__global__ void xmpReorder_compact_kernel(ReorderParams params) {
  uint32_t count=           params.count;
  uint32_t *in_index=       params.in_indices;
  uint32_t in_index_count=  params.in_indices_count;
  uint32_t in_count=        params.in_count;
  uint32_t *in_limbs=       params.in_limbs;
  uint32_t  in_words=       params.in_words;
  uint32_t *out_index=      params.out_indices;
  uint32_t out_index_count= params.out_indices_count;
  uint32_t *out_limbs=      params.out_limbs;
  uint32_t  out_words=      params.out_words;


  for(int idx=blockIdx.y*blockDim.y+threadIdx.y;idx<count;idx+=blockDim.y*gridDim.y) {
    for(int j=blockIdx.x*blockDim.x+threadIdx.x;j<out_words;j+=blockDim.x*gridDim.x) {
      uint32_t i= (NULL==in_index) ? idx : in_index[idx%in_index_count]%in_count;
      uint32_t o= (NULL==out_index) ? idx : out_index[idx%out_index_count];
      //read with indirection
      uint32_t val = (j<in_words) ? in_limbs[i*in_words+j] : 0;
      //write with indirection
      out_limbs[o*out_words+j]=val;
    }
  }
}


xmpError_t XMPAPI xmpIntegersSet(xmpHandle_t handle, xmpIntegers_t out, xmpIntegers_t in, uint32_t count) {
  xmpError_t error=xmpIntegersSetAsync(handle,out,in,count);
  if(error!=xmpErrorSuccess)
    return error;
  XMP_SET_DEVICE(handle);
  hipError_t cerror=hipStreamSynchronize(handle->stream);
  if(cerror==hipSuccess)
    return xmpErrorSuccess;
  else
    return xmpErrorCuda;
}
//copy count integers of size bytes from in into out
xmpError_t XMPAPI xmpIntegersSetAsync(xmpHandle_t handle, xmpIntegers_t out, xmpIntegers_t in, uint32_t count) {
  //verify handle device and in or out device match
  int32_t device=handle->device;
  if(in->device!=device && out->device!=device)
    return xmpErrorInvalidDevice;

  XMP_SET_DEVICE(handle);

  if(in->precision!=out->precision)
    return xmpErrorInvalidPrecision;

  if(out->count<count || in->count<count)
    return xmpErrorInvalidCount;
  
  size_t bytes=in->count*in->nlimbs*sizeof(xmpLimb_t);
  size_t sbytes=in->count*in->stride*sizeof(xmpLimb_t);
  
  xmpExecutionPolicy_t policy=handle->policy;

  if(policy->indices[0]==NULL && policy->indices[1]==NULL && in->count == out->count) {
    xmpFormat_t format=in->getFormat();

    switch(format) {
      case xmpFormatCompact:
        if(hipSuccess!=hipMemcpyAsync(out->climbs,in->climbs,bytes,hipMemcpyDefault,handle->stream))
          return xmpErrorCuda;
        break;
      case xmpFormatStrided:
        if(hipSuccess!=hipMemcpyAsync(out->slimbs,in->slimbs,sbytes,hipMemcpyDefault,handle->stream))
          return xmpErrorCuda;
        break;
      case xmpFormatBoth:
        if(hipSuccess!=hipMemcpyAsync(out->climbs,in->climbs,bytes,hipMemcpyDefault,handle->stream))
          return xmpErrorCuda;
        if(hipSuccess!=hipMemcpyAsync(out->slimbs,in->slimbs,sbytes,hipMemcpyDefault,handle->stream))
          return xmpErrorCuda;
        break;
      case xmpFormatNone:
        return xmpErrorInvalidFormat;
    }
    out->setFormat(format);
  } else {
    //currently only support working in compact format
    in->requireFormat(handle,xmpFormatCompact);

    //create scratch memory (handles, different devices and in-place transfers)
    xmpError_t e=xmpSetNecessaryScratchSize(handle,in->nlimbs*count*sizeof(xmpLimb_t));
    if(e!=xmpErrorSuccess) return e;
    uint32_t *dst=(uint32_t*)handle->scratch;

    //copy to temporary memory on target device
    if(hipSuccess!=hipMemcpyAsync(dst, in->climbs, bytes, hipMemcpyDefault, handle->stream)) {
      return xmpErrorCuda;
    }

    ReorderParams params;

    params.out_limbs=out->climbs;
    params.out_indices=policy->indices[0];
    params.out_indices_count=policy->indices_count[0];
    params.in_limbs=dst;
    params.in_indices=policy->indices[1];
    params.in_indices_count=policy->indices_count[1];
    params.in_count=in->count;
    params.out_words=out->nlimbs;
    params.in_words=in->nlimbs;
    params.count=count;
    
    dim3 blocks, threads;

    threads.x=MIN(out->nlimbs,128);       //1 thread per limb, max of 128
    threads.y=DIV_ROUND_UP(128,threads.x);  //target block size is 128, fill in block size with instances if necessary
    blocks.x=DIV_ROUND_UP(out->nlimbs,threads.x);  //remaining limbs get covered in multiple blocks
    blocks.y=DIV_ROUND_UP(out->count,threads.y);    //handle remainig instances with more blocks

    xmpReorder_compact_kernel<<<blocks,threads>>>(params);

    out->setFormat(xmpFormatCompact);
  }

  XMP_CHECK_CUDA();
  return xmpErrorSuccess;
}

//x along N
//y along limbs
__global__ void xmpC2S_kernel(uint32_t N, uint32_t limbs, uint32_t stride, const uint32_t * in, uint32_t * out) {
  //outer dimension = N
  //inner dimension = limbs
  
  //read strided in inner dimension`
  //write coalesced in outer dimension
  for(uint32_t i=blockIdx.x*blockDim.x+threadIdx.x;i<N;i+=blockDim.x*gridDim.x) {
    for(uint32_t j=blockIdx.y*blockDim.y+threadIdx.y;j<limbs;j+=blockDim.y*gridDim.y) {
      out[j*stride + i] = in[i*limbs + j];
    }
  }
}

//x along limbs
//y along N
__global__ void xmpS2C_kernel(uint32_t N, uint32_t limbs, uint32_t stride, const uint32_t * in, uint32_t * out) {
  //outer dimension = limbs
  //inner dimension = N

  //read strided in inner dimension
  //write coalesced in outer dimension
  for(uint32_t i=blockIdx.x*blockDim.x+threadIdx.x;i<limbs;i+=blockDim.x*gridDim.x) {
    for(uint32_t j=blockIdx.y*blockDim.y+threadIdx.y;j<N;j+=blockDim.y*gridDim.y) {
      out[j*limbs + i] = in[i*stride + j];
    }
  }
}


